#include "hip/hip_runtime.h"
#include "cudaMatMul.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

__global__ void test_kernel(void) { printf("hello from gpu\n"); }

namespace Wrapper {
void wrapper(void) {
  test_kernel<<<1, 5>>>();
  hipDeviceSynchronize();
}
} // namespace Wrapper
